#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <vector>
#include <chrono>

#include "parameters.h"

/*

YTZ Notes:

This kernel implements the ANI-1 featurization scheme. It can process about 3.6 million samples/minute

*/

inline __device__ float dist_diff(float dx, float dy, float dz) {

    return __fsqrt_rn(dx*dx+dy*dy+dz*dz);

}

inline __device__ float f_C(float r_ij, float r_c) {
    if (r_ij <= r_c) {
        return 0.5 * cosf((M_PI * r_ij) / r_c) + 0.5;
    } else {
        return 0;
    }
}

// Linearizes the diagonal-inclusive upper right
// triangle of the symmetric ranks 0 and 1 of a rank-4 tensor
// into a linear index 
inline __device__ int linearize(int i, int j, int k, int l, AniParams params) {
    if(j < i) {
        float tmp = i;
        i = j;
        j = tmp;
    }

    const auto N = params.max_types;
    const auto K = params.Num_A_thetas;
    const auto L = params.Num_A_Rs;

    int basis = (N*(N-1)/2 - (N-i) * (N-i-1)/2 +j);
    
    return basis*K*L + k*L + l;
}


__global__ void inverse(
    const int *sort_idxs,
    int *gather_idxs,
    size_t n_elems) {
    int elem_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(elem_idx < n_elems) {
        gather_idxs[sort_idxs[elem_idx]] = elem_idx;        
    }
} 

__global__ void scatter(
    const int *sorted_global_idxs,
    const int *sorted_local_idxs,
    int *scatter_idxs,
    size_t n_elems) {
    int elem_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(elem_idx < n_elems) {
        scatter_idxs[sorted_global_idxs[elem_idx]] = sorted_local_idxs[elem_idx];
    }
}

// Remind yutong to document what these pointers are.
__global__ void featurize_gpu(
    const float *Xs,
    const float *Ys,
    const float *Zs,

    const int *atomic_nums,
    const int *mol_offsets,
    const int *mol_atom_count,
    const int num_mols, // actually equal to blockDim.x

    const int *scatter_idxs, // LOCAL WITHIN THE ATOM TYPE

    float *X_feat_out_H,
    float *X_feat_out_C,
    float *X_feat_out_N,
    float *X_feat_out_O,
    AniParams params) {

    int mol_idx = blockIdx.x;
    int num_atoms = mol_atom_count[blockIdx.x];
    int block_size = blockDim.x;
    int num_warps = (num_atoms + block_size - 1)/block_size; // how many warps we need to process

    const size_t angular_feature_size = params.Num_A_Rs * params.Num_A_thetas * (params.max_types * (params.max_types+1) / 2);
    const size_t radial_feature_size = params.max_types * params.Num_R_Rs;
    const size_t total_feature_size = angular_feature_size + radial_feature_size;

    for(int warp_idx = 0; warp_idx < num_warps; warp_idx++) {

        int local_atom_idx = warp_idx*block_size + threadIdx.x; // local_local_atom_idx

        if (local_atom_idx >= num_atoms) {
            return;
        }

        // todo: cache into shared mem
        // load all the x y z coordinates
        int g_atom_idx_i = mol_offsets[mol_idx]+local_atom_idx;

        int g_atomic_num_i = atomic_nums[g_atom_idx_i];

        float i_x = Xs[g_atom_idx_i];
        float i_y = Ys[g_atom_idx_i];
        float i_z = Zs[g_atom_idx_i];


        // printf("%d %d %d (%f, %f, %f)\n", mol_idx, local_atom_idx, num_atoms, i_x, i_y, i_z);

        // float *X_feat_i = X_feat_out + scatter_idxs[g_atom_idx_i];

        // if(Atom)

        float *X_feat_out_i;
        if(g_atomic_num_i == 0) {
            X_feat_out_i = X_feat_out_H;
        } else if(g_atomic_num_i == 1) {
            X_feat_out_i = X_feat_out_C;
        } else if(g_atomic_num_i == 2) {
            X_feat_out_i = X_feat_out_N;
        } else {
            X_feat_out_i = X_feat_out_O;
        }
      
        float *radial_feature_buffer_i = X_feat_out_i + scatter_idxs[g_atom_idx_i]*total_feature_size + 0;
        float *angular_feature_buffer_i = X_feat_out_i + scatter_idxs[g_atom_idx_i]*total_feature_size + radial_feature_size;

        for(int j=0; j < num_atoms; j++) {

            int g_atom_idx_j = mol_offsets[mol_idx]+j;
            int g_atomic_num_j = atomic_nums[g_atom_idx_j];

            float j_x = Xs[g_atom_idx_j];
            float j_y = Ys[g_atom_idx_j];
            float j_z = Zs[g_atom_idx_j];

            float d_ij_x = i_x - j_x;
            float d_ij_y = i_y - j_y;
            float d_ij_z = i_z - j_z;

            // printf("(%f, %f, %f)\n", d_ij_x, d_ij_y, d_ij_z);

            float r_ij = dist_diff(d_ij_x, d_ij_y, d_ij_z);

            // float *X_feat_j = X_feat_out + scatter_idxs[g_atom_idx_j];

            float *X_feat_out_j;
            if(g_atomic_num_j == 0) {
                X_feat_out_j = X_feat_out_H;
            } else if(g_atomic_num_j == 1) {
                X_feat_out_j = X_feat_out_C;
            } else if(g_atomic_num_j == 2) {
                X_feat_out_j = X_feat_out_N;
            } else {
                X_feat_out_j = X_feat_out_O;
            }

            float *radial_feature_buffer_j = X_feat_out_j + scatter_idxs[g_atom_idx_j]*total_feature_size + 0;
            // float *radial_feature_buffer_j = X_feat_j + g_atom_idx_j*TOTAL_FEATURE_SIZE + 0;
            // float *angular_feature_buffer_j = X_feat_out + g_atom_idx_j*TOTAL_FEATURE_SIZE + RADIAL_FEATURE_SIZE;


            // if(g_atom_idx_i == 0) {
                // printf("gpu j %d %f\n", j, r_ij);
                // printf("summand, offset, %f, %d\n", summand, scatter_idxs[g_atom_idx_i]*TOTAL_FEATURE_SIZE + atomic_nums[g_atom_idx_j] * params.Num_R_Rs + r_idx);
                // printf("summand, offset, %f, %d\n", summand, scatter_idxs[g_atom_idx_i]*TOTAL_FEATURE_SIZE + atomic_nums[g_atom_idx_j] * params.Num_R_Rs + r_idx);
            // }

            // radial features
            if(r_ij < params.R_Rc && local_atom_idx < j) {
                for(int r_idx = 0; r_idx < params.Num_R_Rs; r_idx++) {
                    float diff = r_ij - params.R_Rs[r_idx];
                    float summand = __expf(-params.R_eta * diff * diff) * f_C(r_ij, params.R_Rc);

                    // // float inner = powf(r_ij - params.R_Rs[r_idx], 2);


                    // // if(isnan(inner) || isinf(inner)) {
                    // //     printf("SUMMAND INNER NAN\n");
                    // // }



                    // float diff = r_ij - params.R_Rs[r_idx];

                    // float inner = diff*diff;


                    // if(isnan(inner) || isinf(inner)) {
                    //     printf("SUMMAND INNER NAN %f %f %f\n", r_ij,  params.R_Rs[r_idx], inner);
                    // }

                    // float a = __expf(-params.R_eta * inner);
                    // float b = f_C(r_ij, params.R_Rc);

                    // if(isnan(a) || isinf(a)) {
                    //     printf("SUMMAND A NAN\n");
                    // }

                    // if(isnan(b) || isinf(b)) {
                    //     printf("SUMMAND B NAN\n");
                    // }


                    // float summand = a*b;


                    // exploit symmetry of the atomic adds
                    auto res1 = atomicAdd(radial_feature_buffer_i + atomic_nums[g_atom_idx_j] * params.Num_R_Rs + r_idx, summand);
                    auto res2 = atomicAdd(radial_feature_buffer_j + atomic_nums[g_atom_idx_i] * params.Num_R_Rs + r_idx, summand);

                    if(isnan(res1) || isinf(res1)) {
                       printf("WTF RADIAL RES1 NAN/INF, offset, %f, %f\n", res1, summand);
                        // : %d, %d, %d, r_ij, r_ik, %f, %f, top %f, bottom %f, i_coords:(%f, %f, %f), j_coords(%f, %f, %f), k_coords(%f, %f, %f)\n",
                            // g_atom_idx_i, g_atom_idx_j, g_atom_idx_k, r_ij, r_ik, d_ij_x*d_ik_x + d_ij_y*d_ik_y + d_ij_z*d_ik_z, r_ij * r_ik, i_x, i_y, i_z, j_x, j_y, j_z, k_x, k_y, k_z);
                    }

                    if(isnan(res2) || isinf(res2)) {
                       printf("WTF RADIAL RES2 NAN/INF, offset, %f, %f\n", res2, summand);
                        // : %d, %d, %d, r_ij, r_ik, %f, %f, top %f, bottom %f, i_coords:(%f, %f, %f), j_coords(%f, %f, %f), k_coords(%f, %f, %f)\n",
                            // g_atom_idx_i, g_atom_idx_j, g_atom_idx_k, r_ij, r_ik, d_ij_x*d_ik_x + d_ij_y*d_ik_y + d_ij_z*d_ik_z, r_ij * r_ik, i_x, i_y, i_z, j_x, j_y, j_z, k_x, k_y, k_z);
                    }

                }
            }

            float A_f_C_ij = f_C(r_ij, params.A_Rc);


            if(r_ij < params.A_Rc) {

                for(size_t k=j+1; k < num_atoms; k++) {

                    if(local_atom_idx == j || local_atom_idx == k || j == k) {
                        continue;
                    }

                    int g_atom_idx_k = mol_offsets[mol_idx]+k;

                    const int an_j = atomic_nums[g_atom_idx_j];
                    const int an_k = atomic_nums[g_atom_idx_k];

                    float k_x = Xs[g_atom_idx_k];
                    float k_y = Ys[g_atom_idx_k];
                    float k_z = Zs[g_atom_idx_k];

                    float d_ik_x = i_x - k_x;
                    float d_ik_y = i_y - k_y;
                    float d_ik_z = i_z - k_z;

                    float r_ik = dist_diff(d_ik_x, d_ik_y, d_ik_z);

                    if(r_ik < params.A_Rc) {

                        // TODO(YTZ): replace with arctan2 trick

                        float inner = (d_ij_x*d_ik_x + d_ij_y*d_ik_y + d_ij_z*d_ik_z) / (r_ij * r_ik);
                        inner = fmaxf(inner, -1.0);
                        inner = fminf(inner,  1.0);

                        // printf("INNER %f\n", inner);

                        float theta_ijk = acosf(inner);

                        // super useful debug
                        if(isnan(theta_ijk) || isinf(theta_ijk)) {
                            printf("WTF NAN/INF: %d, %d, %d, r_ij, r_ik, %f, %f, top %f, bottom %f, i_coords:(%f, %f, %f), j_coords(%f, %f, %f), k_coords(%f, %f, %f)\n",
                                g_atom_idx_i, g_atom_idx_j, g_atom_idx_k, r_ij, r_ik, d_ij_x*d_ik_x + d_ij_y*d_ik_y + d_ij_z*d_ik_z, r_ij * r_ik, i_x, i_y, i_z, j_x, j_y, j_z, k_x, k_y, k_z);
                        }

                        // printf("gpu tijk %d %d %d %f\n", local_atom_idx, j, k, theta_ijk);
                        float A_f_C_ik = f_C(r_ik, params.A_Rc);
                        for(int t=0; t < params.Num_A_thetas; t++) {
                            for(int s=0; s < params.Num_A_Rs; s++) {
                                // (TODO: ytz) do 2*(1-A_Zeta) at the end
                                float inner = (r_ij + r_ik)/2 - params.A_Rs[s]; // powf(x,y) is numerically unstable for negative xs so we simply double them.
                                float summand = powf(2, 1-params.A_zeta) * powf(1+cosf(theta_ijk - params.A_thetas[t]), params.A_zeta) * expf(-params.A_eta*inner*inner) * A_f_C_ij * A_f_C_ik;
                                // printf("summand: %f, \n", summand);
                                // printf("scatter_idxs[g_atom_idx_i]: %d, linearize: %d\n", scatter_idxs[g_atom_idx_i], linearize(an_j, an_k, t, s));
                                // printf("i,j,k,t,s %d %d %d %d %d %d\n", g_atom_idx_i, g_atom_idx_j, g_atom_idx_k, at_idx, ar_idx, linearize(j_type, k_type, at_idx, ar_idx))
                                auto res = atomicAdd(angular_feature_buffer_i + linearize(an_j, an_k, t, s, params), summand);

                                // if(isnan(res) || isinf(res)) {
                                //     printf("WTF ANGULAR SUMMAND NAN/INF: %d, %d, %d, r_ij, r_ik, %f, %f, top %f, bottom %f, i_coords:(%f, %f, %f), j_coords(%f, %f, %f), k_coords(%f, %f, %f)\n",
                                //         g_atom_idx_i, g_atom_idx_j, g_atom_idx_k, r_ij, r_ik, d_ij_x*d_ik_x + d_ij_y*d_ik_y + d_ij_z*d_ik_z, r_ij * r_ik, i_x, i_y, i_z, j_x, j_y, j_z, k_x, k_y, k_z);
                                // }
                            }
                        }
                    }
                }
            } // end radial
        } // end current warp
    } // end all warps

}

// template<typename X_type, typename Y_type>
__device__ static inline float square(float x) {
    return x*x;
}

#include "assert.h"

// Remind yutong to document what these pointers are.
__global__ void featurize_grad_gpu(
    const float *Xs,
    const float *Ys,
    const float *Zs,

    const int *atomic_nums,
    const int *mol_offsets,
    const int *mol_atom_count,
    const int num_mols, // actually equal to blockDim.x

    const int *scatter_idxs, // LOCAL WITHIN THE ATOM TYPE

    const float *H_grads,
    const float *C_grads,
    const float *N_grads,
    const float *O_grads,

    float *X_grads,
    float *Y_grads,
    float *Z_grads,

    AniParams params) {

    int mol_idx = blockIdx.x;
    int num_atoms = mol_atom_count[blockIdx.x];
    int block_size = blockDim.x;
    int num_warps = (num_atoms + block_size - 1)/block_size; // how many warps we need to process

    const size_t angular_feature_size = params.Num_A_Rs * params.Num_A_thetas * (params.max_types * (params.max_types+1) / 2);
    const size_t radial_feature_size = params.max_types * params.Num_R_Rs;
    const size_t total_feature_size = angular_feature_size + radial_feature_size;

    for(int warp_idx = 0; warp_idx < num_warps; warp_idx++) {

        int local_atom_idx = warp_idx*block_size + threadIdx.x; // local_local_atom_idx

        if (local_atom_idx >= num_atoms) {
            return;
        }

        // todo: cache into shared mem
        // load all the x y z coordinates
        int g_atom_idx_i = mol_offsets[mol_idx]+local_atom_idx;
        int g_atomic_num_i = atomic_nums[g_atom_idx_i];

        float i_x = Xs[g_atom_idx_i];
        float i_y = Ys[g_atom_idx_i];
        float i_z = Zs[g_atom_idx_i];

        const float *output_buffer_i;
        if(g_atomic_num_i == 0) {
            output_buffer_i = H_grads;
        } else if(g_atomic_num_i == 1) {
            output_buffer_i = C_grads;
        } else if(g_atomic_num_i == 2) {
            output_buffer_i = N_grads;
        } else {
            output_buffer_i = O_grads;
        }
      
        const float *radial_feature_buffer_i = output_buffer_i + scatter_idxs[g_atom_idx_i]*total_feature_size + 0;
        const float *angular_feature_buffer_i = output_buffer_i + scatter_idxs[g_atom_idx_i]*total_feature_size + radial_feature_size;

        for(int j=0; j < num_atoms; j++) {

            int g_atom_idx_j = mol_offsets[mol_idx]+j;
            // int g_atomic_num_j = atomic_nums[g_atom_idx_j];

            float j_x = Xs[g_atom_idx_j];
            float j_y = Ys[g_atom_idx_j];
            float j_z = Zs[g_atom_idx_j];

            float d_ij_x = i_x - j_x;
            float d_ij_y = i_y - j_y;
            float d_ij_z = i_z - j_z;

            // printf("(%f, %f, %f)\n", d_ij_x, d_ij_y, d_ij_z);

            float r_ij = dist_diff(d_ij_x, d_ij_y, d_ij_z);

            // radial features
            if(r_ij < params.R_Rc && local_atom_idx != j) {
                for(int r_idx = 0; r_idx < params.Num_R_Rs; r_idx++) {

                    float d_y_i = radial_feature_buffer_i[atomic_nums[g_atom_idx_j] * params.Num_R_Rs + r_idx]; // gradient broadcast
                    float lhs_mult = -2*params.R_eta*(-params.R_Rs[r_idx] + r_ij)*(0.5*__cosf(M_PI*r_ij/params.R_Rc) + 0.5)*__expf(-params.R_eta*square(-params.R_Rs[r_idx] + r_ij))/r_ij;
                    float rhs_mult = 0.5*M_PI*__expf(-params.R_eta*square(-params.R_Rs[r_idx] + r_ij))*__sinf(M_PI*r_ij/params.R_Rc)/(params.R_Rc*r_ij);

                    float accum_i_x = (i_x - j_x)*lhs_mult - (i_x - j_x)*rhs_mult;
                    float accum_i_y = (i_y - j_y)*lhs_mult - (i_y - j_y)*rhs_mult;
                    float accum_i_z = (i_z - j_z)*lhs_mult - (i_z - j_z)*rhs_mult;

                    float accum_j_x = (-i_x + j_x)*lhs_mult - (-i_x + j_x)*rhs_mult;
                    float accum_j_y = (-i_y + j_y)*lhs_mult - (-i_y + j_y)*rhs_mult;
                    float accum_j_z = (-i_z + j_z)*lhs_mult - (-i_z + j_z)*rhs_mult;

                    // accumulate locally
                    atomicAdd(X_grads+g_atom_idx_i, accum_i_x * d_y_i);
                    atomicAdd(Y_grads+g_atom_idx_i, accum_i_y * d_y_i);
                    atomicAdd(Z_grads+g_atom_idx_i, accum_i_z * d_y_i);

                    atomicAdd(X_grads+g_atom_idx_j, accum_j_x * d_y_i);
                    atomicAdd(Y_grads+g_atom_idx_j, accum_j_y * d_y_i);
                    atomicAdd(Z_grads+g_atom_idx_j, accum_j_z * d_y_i);

                }
            }

            float A_f_C_ij = f_C(r_ij, params.A_Rc);


            if(r_ij < params.A_Rc) {
                for(size_t k=j+1; k < num_atoms; k++) {
                    if(local_atom_idx == j || local_atom_idx == k || j == k) {
                        continue;
                    }

                    int g_atom_idx_k = mol_offsets[mol_idx]+k;

                    const int an_j = atomic_nums[g_atom_idx_j];
                    const int an_k = atomic_nums[g_atom_idx_k];

                    float k_x = Xs[g_atom_idx_k];
                    float k_y = Ys[g_atom_idx_k];
                    float k_z = Zs[g_atom_idx_k];

                    float d_ik_x = i_x - k_x;
                    float d_ik_y = i_y - k_y;
                    float d_ik_z = i_z - k_z;

                    float r_ik = dist_diff(d_ik_x, d_ik_y, d_ik_z);


                    if(r_ik < params.A_Rc) {
                        for(int t=0; t < params.Num_A_thetas; t++) {
                            for(int s=0; s < params.Num_A_Rs; s++) {

                                float d_y_i = angular_feature_buffer_i[linearize(an_j, an_k, t, s, params)];

                                float dx_ij = i_x - j_x;
                                float dy_ij = i_y - j_y;
                                float dz_ij = i_z - j_z;

                                float dx_ik = i_x - k_x;
                                float dy_ik = i_y - k_y;
                                float dz_ik = i_z - k_z;


                                float d2ij = dx_ij*dx_ij+dy_ij*dy_ij+dz_ij*dz_ij;
                                float d2ik = dx_ik*dx_ik+dy_ik*dy_ik+dz_ik*dz_ik;

                                float dij = __fsqrt_rn(d2ij);
                                float dik = __fsqrt_rn(d2ik);
                                float ijk_swizzle = (dx_ij)*(dx_ik) + (dy_ij)*(dy_ik) + (dz_ij)*(dz_ik);

                                float dijik = dij*dik;
                                float dtheta = __cosf(params.A_thetas[t] - acosf((ijk_swizzle)/(dijik))) + 1;

                                float cos_theta = (ijk_swizzle)/(dijik);

                                float eps = 1e-6;

                                //skipping if 0 or pi
                                if((fabsf(cos_theta - 1) < eps) || (fabsf(cos_theta + 1) < eps)) {
                                    continue;
                                }

                                float p2paz = powf(2.0, -params.A_zeta + 1);

                                // float pfdtpaz = powf(dtheta, params.A_zeta);
                                float pfdtpazn1 = powf(dtheta, params.A_zeta-1);
                                float pfdtpaz = pfdtpazn1*dtheta;
                                float halfcosf_dij = 0.5*__cosf(M_PI*dij/params.A_Rc) + 0.5;
                                float halfcosf_dik = 0.5*__cosf(M_PI*dik/params.A_Rc) + 0.5;
                                float expfpa = __expf(-params.A_eta*powf(-params.A_Rs[s] + (0.5)*dij + (0.5)*dik, 2));
                                float sm_dij = __sinf(M_PI*dij/params.A_Rc);
                                float sm_dik = __sinf(M_PI*dik/params.A_Rc);

                                float full_dijk = halfcosf_dij*halfcosf_dik;

                                float d3ijdik = d2ij*dijik;
                                float dijd3ik = d2ik*dijik;
                                float sinf2a = __sinf(params.A_thetas[t] - acosf(cos_theta));
                                float sqrtfd2ijkpow = __fsqrt_rn(-powf(ijk_swizzle, 2)/((d2ij)*(d2ik)) + 1);
                                // float full_dijk = (halfcosf_dij)*(halfcosf_dik);

                                float pqijk34 = -p2paz*params.A_eta*full_dijk*pfdtpaz*(-params.A_Rs[s] + (0.5)*dij + (0.5)*dik)*expfpa;
                                float fijkl_last = full_dijk*pfdtpazn1*expfpa*sinf2a/(sqrtfd2ijkpow);
                                float trip3p = p2paz*params.A_zeta*fijkl_last;
                                float fma3d = 0.5*p2paz*M_PI*pfdtpaz*expfpa*sm_dij/(params.A_Rc*dij);
                                float fma3e = 0.5*p2paz*M_PI*pfdtpaz*expfpa*sm_dik/(params.A_Rc*dik);

                                float accum_i_x = (dx_ij/dij + dx_ik/dik)*pqijk34 - (-dx_ij*(ijk_swizzle)/(d3ijdik) + -dx_ik*(ijk_swizzle)/(dijd3ik) + (dx_ij + dx_ik)/(dijik))*trip3p - dx_ij*(halfcosf_dik)*fma3d - dx_ik*(halfcosf_dij)*fma3e;
                                float accum_i_y = (dy_ij/dij + dy_ik/dik)*pqijk34 - (-dy_ij*(ijk_swizzle)/(d3ijdik) + -dy_ik*(ijk_swizzle)/(dijd3ik) + (dy_ij + dy_ik)/(dijik))*trip3p - dy_ij*(halfcosf_dik)*fma3d - dy_ik*(halfcosf_dij)*fma3e;
                                float accum_i_z = (dz_ij/dij + dz_ik/dik)*pqijk34 - (-dz_ij*(ijk_swizzle)/(d3ijdik) + -dz_ik*(ijk_swizzle)/(dijd3ik) + (dz_ij + dz_ik)/(dijik))*trip3p - dz_ij*(halfcosf_dik)*fma3d - dz_ik*(halfcosf_dij)*fma3e;

                                float accum_j_x = -dx_ij*pqijk34/dij - p2paz*params.A_zeta*(-dx_ik/(dijik) + dx_ij*(ijk_swizzle)/(d3ijdik))*fijkl_last - -dx_ij*(halfcosf_dik)*fma3d;
                                float accum_j_y = -dy_ij*pqijk34/dij - p2paz*params.A_zeta*(-dy_ik/(dijik) + dy_ij*(ijk_swizzle)/(d3ijdik))*fijkl_last - -dy_ij*(halfcosf_dik)*fma3d;
                                float accum_j_z = -dz_ij*pqijk34/dij - p2paz*params.A_zeta*(-dz_ik/(dijik) + dz_ij*(ijk_swizzle)/(d3ijdik))*fijkl_last - -dz_ij*(halfcosf_dik)*fma3d; 

                                float accum_k_x = -dx_ik*pqijk34/dik - p2paz*params.A_zeta*(-dx_ij/(dijik) + dx_ik*(ijk_swizzle)/(dijd3ik))*fijkl_last - -dx_ik*(halfcosf_dij)*fma3e;
                                float accum_k_y = -dy_ik*pqijk34/dik - p2paz*params.A_zeta*(-dy_ij/(dijik) + dy_ik*(ijk_swizzle)/(dijd3ik))*fijkl_last - -dy_ik*(halfcosf_dij)*fma3e;
                                float accum_k_z = -dz_ik*pqijk34/dik - p2paz*params.A_zeta*(-dz_ij/(dijik) + dz_ik*(ijk_swizzle)/(dijd3ik))*fijkl_last - -dz_ik*(halfcosf_dij)*fma3e;

                                if(isnan(accum_i_x) || isinf(accum_i_x)) {
                                    // auto aaa = -powf(2.0, -params.A_zeta + 1)*params.A_eta*((i_x - j_x)/dij);
                                    // auto bbb = (0.5*cosf(M_PI*dij/params.A_Rc) + 0.5)*(0.5*cosf(M_PI*dik/params.A_Rc) + 0.5);
                                    // auto ccc = powf(cosf(params.A_thetas[t] - acosf((ijk_swizzle)/(dij*dik))) + 1, params.A_zeta);
                                    // auto ddd = (-params.A_Rs[s] + (0.5)*dij + (0.5)*dik);
                                    // auto eee = __expf(-params.A_eta*powf(-params.A_Rs[s] + (0.5)*dij + (0.5)*dik, 2));
                                    // auto fff = powf(2.0, -params.A_zeta + 1)*params.A_zeta*(0.5*cosf(M_PI*dij/params.A_Rc) + 0.5)*(0.5*cosf(M_PI*dik/params.A_Rc) + 0.5);

                                    // auto ggg = powf(cosf(params.A_thetas[t] - acosf((ijk_swizzle)/(dij*dik))) + 1, params.A_zeta);

                                    // auto hhh = ((-i_x + j_x)*(ijk_swizzle)/(powf(powf(i_x - j_x, 2) + powf(i_y - j_y, 2) + powf(i_z - j_z, 2), 1.5)*dik) + (-i_x + k_x)*(ijk_swizzle)/(dij*powf(powf(i_x - k_x, 2) + powf(i_y - k_y, 2) + powf(i_z - k_z, 2), 1.5)) + (2*i_x - j_x - k_x)/(dij*dik));

                                    // auto hihi = __expf(-params.A_eta*powf(-params.A_Rs[s] + (0.5)*dij + (0.5)*dik, 2))*sinf(params.A_thetas[t] - acosf((ijk_swizzle)/(dij*dik)));

                                    // auto iii = sqrt(-powf(ijk_swizzle, 2)/((powf(i_x - j_x, 2) + powf(i_y - j_y, 2) + powf(i_z - j_z, 2))*(powf(i_x - k_x, 2) + powf(i_y - k_y, 2) + powf(i_z - k_z, 2))) + 1)*(cosf(params.A_thetas[t] - acosf((ijk_swizzle)/(dij*dik))) + 1);



                                    // auto jjj = 0.5*powf(2.0, -params.A_zeta + 1)*M_PI*(i_x - j_x)*(0.5*cosf(M_PI*dik/params.A_Rc) + 0.5)*powf(cosf(params.A_thetas[t] - acosf((ijk_swizzle)/(dij*dik))) + 1, params.A_zeta);
                                    // auto kkk = __expf(-params.A_eta*powf(-params.A_Rs[s] + (0.5)*dij + (0.5)*dik, 2))*sinf(M_PI*dij/params.A_Rc)/(params.A_Rc*dij);
                                    // auto lll = 0.5*powf(2.0, -params.A_zeta + 1)*M_PI*(i_x - k_x)*(0.5*cosf(M_PI*dij/params.A_Rc) + 0.5)*powf(cosf(params.A_thetas[t] - acosf((ijk_swizzle)/(dij*dik))) + 1, params.A_zeta);
                                    // auto mmm = __expf(-params.A_eta*powf(-params.A_Rs[s] + (0.5)*dij + (0.5)*dik, 2))*sinf(M_PI*dik/params.A_Rc)/(params.A_Rc*dik);


                                    // printf("garbage accum_i_x detected! %f %f %f %f %f %f %f %f\n", dtheta, acosf(cos_theta), aaa, bbb, ccc, ccc_denom, ddd, ddd_denom);
                                    // assert(0);
                                }

                                // if(isnan(accum_j_x) || isinf(accum_j_x)) {
                                //     printf("garbage accum_j_x detected! \n");
                                //     assert(0);
                                // }
                                // if(isnan(accum_k_x) || isinf(accum_k_x)) {
                                //     printf("garbage accum_k_x detected! \n");
                                //     assert(0);
                                // }

                                // if(isnan(accum_i_y) || isinf(accum_i_y)) {
                                //     printf("garbage accum_i_y detected! \n");
                                //     assert(0);
                                // }
                                // if(isnan(accum_j_y) || isinf(accum_j_y)) {
                                //     printf("garbage accum_j_y detected! \n");
                                //     assert(0);
                                // }
                                // if(isnan(accum_k_y) || isinf(accum_k_y)) {
                                //     printf("garbage accum_k_y detected! \n");
                                //     assert(0);
                                // }

                                // if(isnan(accum_i_z) || isinf(accum_i_z)) {
                                //     printf("garbage accum_i_z detected! \n");
                                //     assert(0);
                                // }
                                // if(isnan(accum_j_z) || isinf(accum_j_z)) {
                                //     printf("garbage accum_j_z detected! \n");
                                //     assert(0);
                                // }
                                // if(isnan(accum_k_z) || isinf(accum_k_z)) {
                                //     printf("garbage accum_k_z detected! \n");
                                //     assert(0);
                                // }

                                // these are accumulated
                                atomicAdd(X_grads+g_atom_idx_i, accum_i_x * d_y_i);
                                atomicAdd(Y_grads+g_atom_idx_i, accum_i_y * d_y_i);
                                atomicAdd(Z_grads+g_atom_idx_i, accum_i_z * d_y_i);

                                atomicAdd(X_grads+g_atom_idx_j, accum_j_x * d_y_i);
                                atomicAdd(Y_grads+g_atom_idx_j, accum_j_y * d_y_i);
                                atomicAdd(Z_grads+g_atom_idx_j, accum_j_z * d_y_i);

                                atomicAdd(X_grads+g_atom_idx_k, accum_k_x * d_y_i);
                                atomicAdd(Y_grads+g_atom_idx_k, accum_k_y * d_y_i);
                                atomicAdd(Z_grads+g_atom_idx_k, accum_k_z * d_y_i);
                            }
                        }
                    }
                }
            } // end radial
        } // end current warp
    } // end all warps

}

// Remind yutong to document what these pointers are.
__global__ void featurize_grad_inverse_gpu(
    const float *Xs,
    const float *Ys,
    const float *Zs,

    const int *atomic_nums,
    const int *mol_offsets,
    const int *mol_atom_count,
    const int num_mols, // actually equal to blockDim.x

    const int *scatter_idxs, // LOCAL WITHIN THE ATOM TYPE
    const float *X_grads,
    const float *Y_grads,
    const float *Z_grads,

    float *H_grads,
    float *C_grads,
    float *N_grads,
    float *O_grads,

    AniParams params) {

    int mol_idx = blockIdx.x;
    int num_atoms = mol_atom_count[blockIdx.x];
    int block_size = blockDim.x;
    int num_warps = (num_atoms + block_size - 1)/block_size; // how many warps we need to process

    const size_t angular_feature_size = params.Num_A_Rs * params.Num_A_thetas * (params.max_types * (params.max_types+1) / 2);
    const size_t radial_feature_size = params.max_types * params.Num_R_Rs;
    const size_t total_feature_size = angular_feature_size + radial_feature_size;

    for(int warp_idx = 0; warp_idx < num_warps; warp_idx++) {

        int local_atom_idx = warp_idx*block_size + threadIdx.x; // local_local_atom_idx

        if (local_atom_idx >= num_atoms) {
            return;
        }

        // todo: cache into shared mem
        // load all the x y z coordinates
        int g_atom_idx_i = mol_offsets[mol_idx]+local_atom_idx;
        int g_atomic_num_i = atomic_nums[g_atom_idx_i];

        float i_x = Xs[g_atom_idx_i];
        float i_y = Ys[g_atom_idx_i];
        float i_z = Zs[g_atom_idx_i];

        float *output_buffer_i;
        if(g_atomic_num_i == 0) {
            output_buffer_i = H_grads;
        } else if(g_atomic_num_i == 1) {
            output_buffer_i = C_grads;
        } else if(g_atomic_num_i == 2) {
            output_buffer_i = N_grads;
        } else {
            output_buffer_i = O_grads;
        }
      
        float *radial_feature_buffer_i = output_buffer_i + scatter_idxs[g_atom_idx_i]*total_feature_size + 0;
        float *angular_feature_buffer_i = output_buffer_i + scatter_idxs[g_atom_idx_i]*total_feature_size + radial_feature_size;

        for(int j=0; j < num_atoms; j++) {

            int g_atom_idx_j = mol_offsets[mol_idx]+j;
            // int g_atomic_num_j = atomic_nums[g_atom_idx_j];

            float j_x = Xs[g_atom_idx_j];
            float j_y = Ys[g_atom_idx_j];
            float j_z = Zs[g_atom_idx_j];

            float d_ij_x = i_x - j_x;
            float d_ij_y = i_y - j_y;
            float d_ij_z = i_z - j_z;

            // printf("(%f, %f, %f)\n", d_ij_x, d_ij_y, d_ij_z);

            float r_ij = dist_diff(d_ij_x, d_ij_y, d_ij_z);

            // radial features
            if(r_ij < params.R_Rc && local_atom_idx != j) {
                for(int r_idx = 0; r_idx < params.Num_R_Rs; r_idx++) {
                        float lhs_mult = -2*params.R_eta*(-params.R_Rs[r_idx] + r_ij)*(0.5*__cosf(M_PI*r_ij/params.R_Rc) + 0.5)*__expf(-params.R_eta*square(-params.R_Rs[r_idx] + r_ij))/r_ij;
                        float rhs_mult = 0.5*M_PI*__expf(-params.R_eta*square(-params.R_Rs[r_idx] + r_ij))*__sinf(M_PI*r_ij/params.R_Rc)/(params.R_Rc*r_ij);

                        float accum_i_x = (i_x - j_x)*lhs_mult - (i_x - j_x)*rhs_mult;
                        float accum_i_y = (i_y - j_y)*lhs_mult - (i_y - j_y)*rhs_mult;
                        float accum_i_z = (i_z - j_z)*lhs_mult - (i_z - j_z)*rhs_mult;

                        float accum_j_x = (-i_x + j_x)*lhs_mult - (-i_x + j_x)*rhs_mult;
                        float accum_j_y = (-i_y + j_y)*lhs_mult - (-i_y + j_y)*rhs_mult;
                        float accum_j_z = (-i_z + j_z)*lhs_mult - (-i_z + j_z)*rhs_mult;


                        float accumulant = accum_i_x * X_grads[g_atomic_num_i] + accum_i_y * Y_grads[g_atomic_num_i] + accum_i_z * Z_grads[g_atomic_num_i];
                        radial_feature_buffer_i[atomic_nums[g_atom_idx_j] * params.Num_R_Rs + r_idx] += accumulant;


                        accumulant = accum_j_x * X_grads[g_atomic_num_i] + accum_j_y * Y_grads[g_atomic_num_i] + accum_j_z * Z_grads[g_atomic_num_i];
                        atomicAdd(radial_feature_buffer_i + atomic_nums[g_atom_idx_j] * params.Num_R_Rs + r_idx, accumulant);
                }
            }

            float A_f_C_ij = f_C(r_ij, params.A_Rc);


            if(r_ij < params.A_Rc) {
                for(size_t k=j+1; k < num_atoms; k++) {
                    if(local_atom_idx == j || local_atom_idx == k || j == k) {
                        continue;
                    }

                    int g_atom_idx_k = mol_offsets[mol_idx]+k;

                    const int an_j = atomic_nums[g_atom_idx_j];
                    const int an_k = atomic_nums[g_atom_idx_k];

                    float k_x = Xs[g_atom_idx_k];
                    float k_y = Ys[g_atom_idx_k];
                    float k_z = Zs[g_atom_idx_k];

                    float d_ik_x = i_x - k_x;
                    float d_ik_y = i_y - k_y;
                    float d_ik_z = i_z - k_z;

                    float r_ik = dist_diff(d_ik_x, d_ik_y, d_ik_z);

                    if(r_ik < params.A_Rc) {
                        for(int t=0; t < params.Num_A_thetas; t++) {
                            for(int s=0; s < params.Num_A_Rs; s++) {
                                float dx_ij = i_x - j_x;
                                float dy_ij = i_y - j_y;
                                float dz_ij = i_z - j_z;

                                float dx_ik = i_x - k_x;
                                float dy_ik = i_y - k_y;
                                float dz_ik = i_z - k_z;


                                float d2ij = dx_ij*dx_ij+dy_ij*dy_ij+dz_ij*dz_ij;
                                float d2ik = dx_ik*dx_ik+dy_ik*dy_ik+dz_ik*dz_ik;

                                float dij = __fsqrt_rn(d2ij);
                                float dik = __fsqrt_rn(d2ik);
                                float ijk_swizzle = (dx_ij)*(dx_ik) + (dy_ij)*(dy_ik) + (dz_ij)*(dz_ik);

                                float dijik = dij*dik;
                                float dtheta = __cosf(params.A_thetas[t] - acosf((ijk_swizzle)/(dijik))) + 1;

                                float cos_theta = (ijk_swizzle)/(dijik);

                                float eps = 1e-6;

                                //skipping if 0 or pi
                                if((fabsf(cos_theta - 1) < eps) || (fabsf(cos_theta + 1) < eps)) {
                                    continue;
                                }

                                float p2paz = powf(2.0, -params.A_zeta + 1);

                                // float pfdtpaz = powf(dtheta, params.A_zeta);
                                float pfdtpazn1 = powf(dtheta, params.A_zeta-1);
                                float pfdtpaz = pfdtpazn1*dtheta;
                                float halfcosf_dij = 0.5*__cosf(M_PI*dij/params.A_Rc) + 0.5;
                                float halfcosf_dik = 0.5*__cosf(M_PI*dik/params.A_Rc) + 0.5;
                                float expfpa = __expf(-params.A_eta*powf(-params.A_Rs[s] + (0.5)*dij + (0.5)*dik, 2));
                                float sm_dij = __sinf(M_PI*dij/params.A_Rc);
                                float sm_dik = __sinf(M_PI*dik/params.A_Rc);

                                float full_dijk = halfcosf_dij*halfcosf_dik;

                                float d3ijdik = d2ij*dijik;
                                float dijd3ik = d2ik*dijik;
                                float sinf2a = __sinf(params.A_thetas[t] - acosf(cos_theta));
                                float sqrtfd2ijkpow = __fsqrt_rn(-powf(ijk_swizzle, 2)/((d2ij)*(d2ik)) + 1);
                                // float full_dijk = (halfcosf_dij)*(halfcosf_dik);

                                float pqijk34 = -p2paz*params.A_eta*full_dijk*pfdtpaz*(-params.A_Rs[s] + (0.5)*dij + (0.5)*dik)*expfpa;
                                float fijkl_last = full_dijk*pfdtpazn1*expfpa*sinf2a/(sqrtfd2ijkpow);
                                float trip3p = p2paz*params.A_zeta*fijkl_last;
                                float fma3d = 0.5*p2paz*M_PI*pfdtpaz*expfpa*sm_dij/(params.A_Rc*dij);
                                float fma3e = 0.5*p2paz*M_PI*pfdtpaz*expfpa*sm_dik/(params.A_Rc*dik);

                                float accum_i_x = (dx_ij/dij + dx_ik/dik)*pqijk34 - (-dx_ij*(ijk_swizzle)/(d3ijdik) + -dx_ik*(ijk_swizzle)/(dijd3ik) + (dx_ij + dx_ik)/(dijik))*trip3p - dx_ij*(halfcosf_dik)*fma3d - dx_ik*(halfcosf_dij)*fma3e;
                                float accum_i_y = (dy_ij/dij + dy_ik/dik)*pqijk34 - (-dy_ij*(ijk_swizzle)/(d3ijdik) + -dy_ik*(ijk_swizzle)/(dijd3ik) + (dy_ij + dy_ik)/(dijik))*trip3p - dy_ij*(halfcosf_dik)*fma3d - dy_ik*(halfcosf_dij)*fma3e;
                                float accum_i_z = (dz_ij/dij + dz_ik/dik)*pqijk34 - (-dz_ij*(ijk_swizzle)/(d3ijdik) + -dz_ik*(ijk_swizzle)/(dijd3ik) + (dz_ij + dz_ik)/(dijik))*trip3p - dz_ij*(halfcosf_dik)*fma3d - dz_ik*(halfcosf_dij)*fma3e;

                                float accum_j_x = -dx_ij*pqijk34/dij - p2paz*params.A_zeta*(-dx_ik/(dijik) + dx_ij*(ijk_swizzle)/(d3ijdik))*fijkl_last - -dx_ij*(halfcosf_dik)*fma3d;
                                float accum_j_y = -dy_ij*pqijk34/dij - p2paz*params.A_zeta*(-dy_ik/(dijik) + dy_ij*(ijk_swizzle)/(d3ijdik))*fijkl_last - -dy_ij*(halfcosf_dik)*fma3d;
                                float accum_j_z = -dz_ij*pqijk34/dij - p2paz*params.A_zeta*(-dz_ik/(dijik) + dz_ij*(ijk_swizzle)/(d3ijdik))*fijkl_last - -dz_ij*(halfcosf_dik)*fma3d; 

                                float accum_k_x = -dx_ik*pqijk34/dik - p2paz*params.A_zeta*(-dx_ij/(dijik) + dx_ik*(ijk_swizzle)/(dijd3ik))*fijkl_last - -dx_ik*(halfcosf_dij)*fma3e;
                                float accum_k_y = -dy_ik*pqijk34/dik - p2paz*params.A_zeta*(-dy_ij/(dijik) + dy_ik*(ijk_swizzle)/(dijd3ik))*fijkl_last - -dy_ik*(halfcosf_dij)*fma3e;
                                float accum_k_z = -dz_ik*pqijk34/dik - p2paz*params.A_zeta*(-dz_ij/(dijik) + dz_ik*(ijk_swizzle)/(dijd3ik))*fijkl_last - -dz_ik*(halfcosf_dij)*fma3e;


                                float accumulant = accum_i_x * X_grads[g_atomic_num_i] + accum_i_y * Y_grads[g_atomic_num_i] + accum_i_z * Z_grads[g_atomic_num_i];
                                atomicAdd(angular_feature_buffer_i + linearize(an_j, an_k, t, s, params), accumulant);

                                accumulant = accum_j_x * X_grads[g_atomic_num_i] + accum_j_y * Y_grads[g_atomic_num_i] + accum_j_z * Z_grads[g_atomic_num_i];
                                atomicAdd(angular_feature_buffer_i + linearize(an_j, an_k, t, s, params), accumulant);

                                accumulant = accum_k_x * X_grads[g_atomic_num_i] + accum_k_y * Y_grads[g_atomic_num_i] + accum_k_z * Z_grads[g_atomic_num_i];
                                atomicAdd(angular_feature_buffer_i + linearize(an_j, an_k, t, s, params), accumulant);
                            }
                        }
                    }
                }
            } // end radial
        } // end current warp
    } // end all warps

}



template<typename T>
T *cudaMallocSimple(size_t n) {
    T *d_obj;

    std::cout << "mallocing:" << n*sizeof(T) << "bytes\n";
    assert(hipMalloc(&d_obj, n*sizeof(T)) == 0);
    return d_obj;
}

template<typename T>
void cudaCopySimple(T *obj, size_t n, T *d_obj) {
    assert(hipMemcpy(d_obj, obj, n*sizeof(T), hipMemcpyHostToDevice) == 0);
}

typedef std::chrono::high_resolution_clock Clock;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}   

// int main(void) {

//     auto X_obj = cnpy::npy_load("Xs.npy");
//     auto Y_obj = cnpy::npy_load("Ys.npy");
//     auto Z_obj = cnpy::npy_load("Zs.npy");
//     auto A_obj = cnpy::npy_load("As.npy");

//     auto MOs = cnpy::npy_load("MOs.npy");
//     auto MACs = cnpy::npy_load("MACs.npy");

//     float *d_Xs = cudaMallocSimple<float>(X_obj.shape[0]);
//     float *d_Ys = cudaMallocSimple<float>(Y_obj.shape[0]);
//     float *d_Zs = cudaMallocSimple<float>(Z_obj.shape[0]);
//     int *d_As = cudaMallocSimple<int>(A_obj.shape[0]);
//     int *d_MOs = cudaMallocSimple<int>(MOs.shape[0]);
//     int *d_MACs = cudaMallocSimple<int>(MACs.shape[0]); // max

//     size_t n_total_atoms = X_obj.shape[0];
//     size_t n_mols  = MOs.shape[0];


//     int sort_num_items = n_total_atoms; // change to upperbound later, max number of atoms per block

//     int *d_vals_in = cudaMallocSimple<int>(sort_num_items);
//     int *sort_idxs = cudaMallocSimple<int>(sort_num_items);
//     int *inv_idxs  = cudaMallocSimple<int>(sort_num_items);

//     std::vector<int> idxs(sort_num_items);

//     for(size_t i=0; i < sort_num_items; i++) {
//         idxs[i] = i;
//     }

//     cudaCopySimple(&idxs[0], sort_num_items, d_vals_in);
//     int *d_keys_out = cudaMallocSimple<int>(sort_num_items);
//     void *d_temp_storage = NULL;
//     size_t temp_storage_bytes = 0;

//     // determine size requirements
//     gpuErrchk(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_As, d_keys_out, d_vals_in, sort_idxs, sort_num_items));
//     gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes)) ;

//     // SETUP DONE

//     float* d_X_feat;

//     printf("Total number of atoms: %d \n", n_total_atoms);

//     std::cout << "mallocing:" << n_total_atoms*TOTAL_FEATURE_SIZE*sizeof(float) << "bytes\n";

//     hipMalloc(&d_X_feat, n_total_atoms*TOTAL_FEATURE_SIZE*sizeof(float)); 

//     auto start = Clock::now();




//     // int i=0;
//     for(size_t i=0; i < 100000; i++) {

//         int num_items = n_total_atoms; // upper bound this to a fixed num

//         std::cout << i << std::endl;    
//         cudaCopySimple(X_obj.data<float>(), X_obj.shape[0], d_Xs);
//         cudaCopySimple(Y_obj.data<float>(), Y_obj.shape[0], d_Ys);
//         cudaCopySimple(Z_obj.data<float>(), Z_obj.shape[0], d_Zs);
//         cudaCopySimple(A_obj.data<int>(), A_obj.shape[0], d_As);

//         cudaCopySimple(MOs.data<int>(), MOs.shape[0], d_MOs);
//         cudaCopySimple(MACs.data<int>(), MACs.shape[0], d_MACs); // max

//         assert(hipMemset(d_X_feat, 0, n_total_atoms*TOTAL_FEATURE_SIZE*sizeof(float)) == 0);

//         // atom type counters
//         std::vector<int> counts(MAX_ATOM_TYPES, 0);

//         for(size_t j=0; j < n_total_atoms; j++) {
//             counts[A_obj.data<int>()[j]] += 1;
//         }


//         // 1. Sort by atom pairs.
//         // 2. 

//         // GPU
//         gpuErrchk(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_As, d_keys_out, d_vals_in, sort_idxs, sort_num_items));
//         inverse<<<n_mols, 32>>>(sort_idxs, inv_idxs, sort_num_items); // invert
//         gpuErrchk(hipPeekAtLastError());



//         // follow up with a segment reduce

//         // std::vector<int> test(sort_num_items);
//         // hipMemcpy(&test[0], inv_idxs, n_total_atoms*sizeof(int), hipMemcpyDeviceToHost);

//         // for(auto v : test) {
//         //     std::cout << v << " ";
//         // }

//         // return;



//         // CPU
//         // std::vector<int> buffer(A_obj.data<int>(), A_obj.data<int>() + A_obj.shape[0]);
//         // std::vector<int> h_sort_idx = sort_indexes(buffer);
//         // for(size_t k=0; k < sort_num_items; k++) {
//         //     buffer[h_sort_idx[k]] = k;
//         // }
//         // cudaCopySimple(&buffer[0], sort_num_items, inv_idxs);


//         //START
//         featurize<<<n_mols, 32>>>(
//             d_Xs,
//             d_Ys,
//             d_Zs,
//             d_As,
//             d_MOs,
//             d_MACs,
//             n_mols,
//             inv_idxs,
//             d_X_feat);


//         gpuErrchk( hipPeekAtLastError() );

//         auto end = Clock::now();
//         auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();


//         std::cout << "DURATION:" << duration << std::endl;

//         std::cout << "samples per minute:" << (float((i+1)*n_mols) / duration) * 60 * 1e9 << std::endl;

//     }

//     std::vector<float> X_feat(n_total_atoms*TOTAL_FEATURE_SIZE, 0);
//     hipMemcpy(&X_feat[0], d_X_feat, n_total_atoms*TOTAL_FEATURE_SIZE*sizeof(int), hipMemcpyDeviceToHost);





// }
